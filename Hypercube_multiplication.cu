#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<>
#include<hip/device_functions.h>
#include<stdio.h>
#include<Windows.h>
#include<string.h>

#define n 2
__device__ int getGlobalIdx_1D_2D() {
	return blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
}

__global__ void hypercubeMultiplication(int *C, const int *A, const int *B) {
	int world_rank = getGlobalIdx_1D_2D();
	int i;
	__shared__ int a[n*n], b[n*n], p[n*n];
	int x[n], y[n], c[n];
	memset(c, 0, n * sizeof(int));
	memset(p, 0, n * n * sizeof(int));
	//initialize
	if(world_rank==0)
	for (i = 0; i<n*n; i++) {
		a[i] = A[i];
		b[i] = B[i];
	}
	__syncthreads();

	//step 3 of the algorithm
	for (i = 0; i < n; i++) {
		y[i] = b[i + threadIdx.y*n];
	
	}
	__syncthreads();

		//step 4 of the algorithm
	for(i = 0; i < n; i++)
		x[i] = a[threadIdx.x*n + i];
	__syncthreads();

	//calculate the product
	for (i = 0; i < n; i++) {
		c[i] = x[i] * y[i];
		p[world_rank] += c[i];

	}
	__syncthreads();

	//send the values to the global memory
	if((world_rank%(n+1))==0)
		for (i = 0; i < n; i++) {
			C[threadIdx.y * n + i] = p[i*n + threadIdx.x];
		}
	__syncthreads();
	
}

int main(int argc, char **argv) {
	int a1[n*n], b11[n*n], b1[n*n], c1[n*n];
	int *A, *B, *C;
	int i,j;

	memset(a1, 0, sizeof(a1));
	memset(b1, 0, sizeof(a1));
	memset(c1, 0, sizeof(a1));

	hipMalloc((void**)&A, sizeof(a1));
	hipMalloc((void**)&B, sizeof(b1));
	hipMalloc((void**)&C, sizeof(c1));

	printf("Enter the values of the matrices A and B:\n");

	for (i = 0; i < n*n; i++) {
		scanf("%d", &a1[i]);
	}
	for (i = 0; i < n*n; i++) {
		scanf("%d", &b11[i]);
	}
	for (i = 0; i < n; i++) {								//transpose the matrix B
			for (j = 0; j < n; j++) {
				b1[i*n + j] = b11[i + j*n];
		}
	}

	hipMemcpy(A, a1, sizeof(a1), hipMemcpyHostToDevice);
	hipMemcpy(B, b1, sizeof(a1), hipMemcpyHostToDevice);

	dim3 nThreads(n, n);
	hypercubeMultiplication << <1, nThreads >> > (C, A, B);

	hipMemcpy(c1, C, sizeof(c1), hipMemcpyDeviceToHost);
	printf("The result is:\n");
	for (i = 0; i < n*n; i++) {
		printf("%d ", c1[i]);
		if ((i + 1) % n == 0)
			printf("\n");
	}

	Sleep(20000);
	hipFree(A);
	hipFree(B);
	hipFree(C);

	return 1;
}